#include "tensor.cuh"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

template <typename scalar_t, int DIMS>
Tensor<scalar_t, DIMS> Tensor<scalar_t, DIMS>::reshape(unsigned[DIMS] new_shape)
{
    Tensor<scalar_t, DIMS> ret(new_shape);
    return ret;
}
