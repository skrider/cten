#include "hip/hip_runtime.h"
#include "gemm.cuh"
#include "hip/hip_runtime_api.h"
#include "tensor.cuh"

template <typename T>
void gemm2(Tensor<T, 2> out, Tensor<T, 2> a, Tensor<T, 2> b, Tensor<T, 2> c,
           T alpha, T beta) {
  out.fill(0);
  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks(c.shape[0] / WARP_SIZE, c.shape[1] / WARP_SIZE);
  Gemm2<<<numBlocks, threadsPerBlock>>>(out, a, b, c, alpha, beta);
}

template <typename T, uint BLOCK_SIZE = 32>
__global__ void Gemm2(Tensor<T, 2> out,     // [rows, cols]
                      const Tensor<T, 2> a, // [rows, inner]
                      const Tensor<T, 2> b, // [inner, cols]
                      const Tensor<T, 2> c, // [rows, inner]
                      const T alpha, const T beta) {
  static __shared__ T b_block[BLOCK_SIZE][BLOCK_SIZE];
  static __shared__ T a_t_block[BLOCK_SIZE][BLOCK_SIZE];

  int c_i[2] = {blockDim.y * blockIdx.y + threadIdx.y,
                blockDim.x * blockIdx.x + threadIdx.x};

  T acc = beta * c(c_i);

  for (int ii = 0; ii < a.shape[1]; ii += BLOCK_SIZE) {
    // fetch block of b
    int b_i[2] = {ii + threadIdx.y, blockDim.x * blockIdx.x + threadIdx.x};
    b_block[threadIdx.y][threadIdx.x] = b(b_i);

    // fetch block of a
    int a_i[2] = {blockDim.y * blockIdx.y + threadIdx.y, ii + threadIdx.x};
    a_t_block[threadIdx.x][threadIdx.y] = a(a_i);

    // init accumulator
    __syncthreads();

#pragma unroll
    for (int i = 0; i < BLOCK_SIZE; i++)
      // use i as the first index to fetch entire line at once
      acc += a_t_block[i][threadIdx.y] * b_block[i][threadIdx.x];
  }
  out(c_i) = alpha * acc;
}

template void gemm2(Tensor<int, 2> out, Tensor<int, 2> a, Tensor<int, 2> b,
                    Tensor<int, 2> c, int alpha, int beta);
