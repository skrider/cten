#include <vector>
#include "tensor.h"
#include "cuda_tensor.h"
#include <initializer_list>
#include <functional>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

template <class T>
CudaTensor<T>::CudaTensor(std::initializer_list<unsigned> shape) : Tensor<T>(shape)
{
    int count = sizeof(T) * Tensor<T>::size();
    checkCudaErrors(hipMalloc(&data, count));
    checkCudaErrors(hipMemset(data, 0, count));
}

template <class T>
CudaTensor<T> &CudaTensor<T>::operator=(const CudaTensor<T> &t)
{
    if (this != &t)
    {
        Tensor<T>::shape_ = t.shape_;
        int count = Tensor<T>::size() * sizeof(T);

        checkCudaErrors(hipFree(data));
        checkCudaErrors(hipMalloc(&data, count));
        checkCudaErrors(hipMemcpy(data, t.data, count, hipMemcpyDeviceToDevice));
    }
    return *this;
}

template <class T>
void CudaTensor<T>::set(std::initializer_list<int> index, T value)
{
    int offset = Tensor<T>::offset(index);
    checkCudaErrors(hipMemcpy(data + offset, &value, sizeof(T), hipMemcpyHostToDevice));
}

template <class T>
T CudaTensor<T>::operator()(std::initializer_list<int> index) const
{
    int offset = Tensor<T>::offset(index);
    T value;
    checkCudaErrors(hipMemcpy(&value, data + offset, sizeof(T), hipMemcpyDeviceToHost));
    return value;
}

template class CudaTensor<int>;
