#include <vector>
#include "tensor.h"
#include "cuda_tensor.h"
#include <vector>
#include <functional>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

template <class T>
CudaTensor<T>::CudaTensor(std::vector<unsigned> shape) : Tensor<T>(shape)
{
    int count = sizeof(T) * Tensor<T>::size();
    checkCudaErrors(hipMalloc(&data, count));
    checkCudaErrors(hipMemset(data, 0, count));
}

template <class T>
CudaTensor<T> &CudaTensor<T>::operator=(const CudaTensor<T> &t)
{
    if (this != &t)
    {
        Tensor<T>::shape_ = t.shape_;
        int count = Tensor<T>::size() * sizeof(T);

        checkCudaErrors(hipFree(data));
        checkCudaErrors(hipMalloc(&data, count));
        checkCudaErrors(hipMemcpy(data, t.data, count, hipMemcpyDeviceToDevice));
    }
    return *this;
}

template <class T>
void CudaTensor<T>::set(std::vector<int> index, T value)
{
    int offset = Tensor<T>::offset(index);
    checkCudaErrors(hipMemcpy(data + offset, &value, sizeof(T), hipMemcpyHostToDevice));
}

template <class T>
T CudaTensor<T>::operator()(std::vector<int> index) const
{
    int offset = Tensor<T>::offset(index);
    T value;
    checkCudaErrors(hipMemcpy(&value, data + offset, sizeof(T), hipMemcpyDeviceToHost));
    return value;
}

template <class T>
CpuTensor<T> *CudaTensor<T>::cpu() const
{
    CpuTensor<T> *ret = new CpuTensor<T>(Tensor<T>::shape_);
    int count = Tensor<T>::size() * sizeof(T);
    checkCudaErrors(hipMemcpy(ret->data.data(), data, count, hipMemcpyDeviceToHost));
    return ret;
}

template class CudaTensor<int>;
