#include "hip/hip_runtime.h"
#include "ops.cuh"
#include "tensor.cuh"
#include "hip/hip_runtime_api.h"

template <typename scalar_t>
Tensor<scalar_t, 2> gemm(
    Tensor<scalar_t, 2> a,
    Tensor<scalar_t, 2> b,
    Tensor<scalar_t, 2> c,
    scalar_t alpha,
    scalar_t beta)
{
    Tensor<scalar_t, 2> out(c.shape());
}

// Gemm for a batch size of one at optimization level 1. Basic matrix multiply.
// Each thread computes one element of output.
template <typename scalar_t>
__global__ void GemmSingle1(
    scalar_t *out,                  // [rows, cols]
    const scalar_t *__restrict__ a, // [rows, inner]
    const scalar_t *__restrict__ b, // [inner, cols]
    const scalar_t *__restrict__ c, // [rows, cols]
    const scalar_t alpha,
    const scalar_t beta,
    int rows,
    int cols,
    int inner)
{
    // offset of the out element from beginning of the array
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= rows || col >= cols)
        return;

    int offset = row * cols + col;

    out[offset] = beta * c[offset];
#pragma unroll
    for (int i = 0; i < inner; i++)
    {
        out[offset] += a[row * inner + i] * b[i * inner + col];
    }
}
