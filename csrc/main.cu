#include "hip/hip_runtime.h"
#include "gemm.cuh"
#include "hip/hip_runtime_api.h"
#include "rand.cuh"
#include "tensor.cuh"
#include "tensor_utils.cuh"
#include "utils.cuh"
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define ROW (1 << 12)
#define COL (1 << 12)
#define INNER (1 << 12)

#define WARMUP 2
#define N 10

int main(int argc, char **argv) {
  int seed = atoi(argv[0]);

  Tensor<float, 2> a({ROW, INNER});
  Tensor<float, 2> b({INNER, COL});
  Tensor<float, 2> c({ROW, COL});
  Tensor<float, 2> out({ROW, COL});
  Tensor<float, 2> out1({ROW, COL});
  float alpha = 1.0f;
  float beta = -1.0f;

  randn(a, 0.0, 1.0, 42);
  randn(b, 0.0, 1.0, 43);
  randn(c, 0.0, 1.0, 44);

  for (int i = 0; i < WARMUP; i++) {
    gemm2(out, a, b, c, alpha, beta);
    gemm3(out, a, b, c, alpha, beta);
  }

  for (int i = 0; i < N; i++) {
    gemm2(out, a, b, c, alpha, beta);
    checkCudaErrors(hipDeviceSynchronize());
  }

  for (int i = 0; i < N; i++) {
    gemm3(out1, a, b, c, alpha, beta);
    checkCudaErrors(hipDeviceSynchronize());
  }

  // TODO floating point error is significant - resolve with FMA
  cout << "pass: " << allclose(out1, out, (float)5e-3) << endl;

  Tensor<float, 2> d = (out - out1);
  cout << "max: " << d.max() << endl;
  auto i = d.argmax();
  cout << i[0] << ", " << i[1] << endl;

  cout << "min: " << d.min() << endl;
  i = d.argmin();
  cout << i[0] << ", " << i[1] << endl;
}
