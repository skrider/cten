#include "gemm.cuh"
#include "hip/hip_runtime_api.h"
#include "rand.cuh"
#include "tensor.cuh"
#include "utils.cuh"
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define ROW (1 << 12)
#define COL (1 << 12)
#define INNER (1 << 12)

#define WARMUP 1
#define N 1

int main(int argc, char **argv) {
  Tensor<float, 2> a({ROW, INNER});
  Tensor<float, 2> b({INNER, COL});
  Tensor<float, 2> c({ROW, COL});
  Tensor<float, 2> out({ROW, COL});
  float alpha = 1.0f;
  float beta = -1.0f;

  randn(a, 0.0, 1.0, 42);
  randn(b, 0.0, 1.0, 42);
  randn(c, 0.0, 1.0, 42);

  for (int i = 0; i < WARMUP; i++) {
    gemm1(out, a, b, c, alpha, beta);
    gemm2(out, a, b, c, alpha, beta);
  }

  for (int i = 0; i < N; i++) {
    gemm1(out, a, b, c, alpha, beta);
    checkCudaErrors(hipDeviceSynchronize());
  }

  cout << "element:      " << out.get({31, 30}) << endl;

  for (int i = 0; i < N; i++) {
    gemm2(out, a, b, c, alpha, beta);
    checkCudaErrors(hipDeviceSynchronize());
  }

  cout << "element:      " << out.get({31, 31}) << endl;
}
