#include "hip/hip_runtime.h"
#include "gemm.cuh"
#include "hip/hip_runtime_api.h"
#include "tensor.cuh"

template <typename scalar_t>
void gemm1(Tensor<scalar_t, 2> out, Tensor<scalar_t, 2> a,
           Tensor<scalar_t, 2> b, Tensor<scalar_t, 2> c, scalar_t alpha,
           scalar_t beta) {
  out.fill(0);
  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks(ROUND(c.shape[0], threadsPerBlock.x) / threadsPerBlock.x,
                 ROUND(c.shape[1], threadsPerBlock.y) / threadsPerBlock.y);
  Gemm1<<<numBlocks, threadsPerBlock>>>(out, a, b, c, alpha, beta);
}

// Gemm for a batch size of one at optimization level 1. Basic matrix multiply.
// Each thread computes one element of output.
template <typename scalar_t>
__global__ void Gemm1(Tensor<scalar_t, 2> out,     // [rows, cols]
                      const Tensor<scalar_t, 2> a, // [rows, inner]
                      const Tensor<scalar_t, 2> b, // [inner, cols]
                      const Tensor<scalar_t, 2> c, // [rows, inner]
                      const scalar_t alpha, const scalar_t beta) {
  // offset of the out element from beginning of the array
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  int c_i[2] = {row, col};
  out(c_i) = beta * c(c_i);
#pragma unroll
  for (int i = 0; i < a.shape[1]; i++) {
    int a_i[2] = {row, i};
    int b_i[2] = {i, col};
    out(c_i) += alpha * a(a_i) * b(b_i);
  }
}

template void gemm1(Tensor<int, 2> out, Tensor<int, 2> a, Tensor<int, 2> b,
                    Tensor<int, 2> c, int alpha, int beta);
