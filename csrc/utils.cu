#include "hip/hip_runtime_api.h"
#include "utils.cuh"
#include <algorithm>
#include <array>
#include <bits/range_access.h>
#include <hip/hip_runtime.h>
#include <iostream>

void printDeviceProperties() {
  hipDeviceProp_t prop;
  int device;

  hipGetDevice(&device); // get the current device
  hipGetDeviceProperties(&prop, device);

  std::cout << "Warp size: " << prop.warpSize << std::endl;
}

template <typename T, uint D> std::array<T, D> packCArr(T *c_array) {
  std::array<T, D> std_array;
  for (int i = 0; i < D; i++)
    std_array[i] = c_array[i];
  return std_array;
}

template std::array<uint, 2> packCArr<uint, 2>(uint *c_array);
