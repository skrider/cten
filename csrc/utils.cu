#include "hip/hip_runtime_api.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "utils.cuh"

void printDeviceProperties()
{
    hipDeviceProp_t prop;
    int device;

    hipGetDevice(&device); // get the current device
    hipGetDeviceProperties(&prop, device);

    std::cout << "Warp size: " << prop.warpSize << std::endl;
}