#include "hip/hip_runtime_api.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "utils.cuh"
#include <array>
#include <algorithm>
#include <bits/range_access.h>

void printDeviceProperties()
{
    hipDeviceProp_t prop;
    int device;

    hipGetDevice(&device); // get the current device
    hipGetDeviceProperties(&prop, device);

    std::cout << "Warp size: " << prop.warpSize << std::endl;
}

template <typename T, uint D>
std::array<T, D> packCArr(T c_array[D])
{
    std::array<T, D> std_array;
    for (int i = 0; i < D; i++)
        std_array[i] = c_array[i];
    return std_array;
}

template std::array<uint, 2> packCArr<uint, 2>(uint c_array[2]);